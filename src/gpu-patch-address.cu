#include "hip/hip_runtime.h"
/*
 * Use C style programming in this file
 */
#include "gpu-patch.h"
#include "gpu-queue.h"
#include "utils.h"

#include <sanitizer_patching.h>

/*
 * Monitor each shared and global memory access.
 */
static 
__device__ __forceinline__
SanitizerPatchResult
memory_access_callback
(
 void *user_data,
 uint64_t pc,
 void *address,
 uint32_t size,
 uint32_t flags,
 const void *new_value
) 
{
  gpu_patch_buffer_t *buffer = (gpu_patch_buffer_t *)user_data;

  // 1. Init values
  uint32_t active_mask = __activemask();
  uint32_t laneid = get_laneid();
  uint32_t first_laneid = __ffs(active_mask) - 1;

  gpu_patch_record_t *record = NULL;
  if (laneid == first_laneid) {
    // 3. Get a record
    gpu_patch_record_t *records = (gpu_patch_record_t *)buffer->records;
    record = records + gpu_queue_get(buffer, buffer->flags & GPU_PATCH_ANALYSIS); 

    // 4. Assign basic values
    record->flags = flags;
    record->active = active_mask;
  }

  __syncwarp(active_mask);

  uint64_t r = (uint64_t)record;
  record = (gpu_patch_record_t *)shfl(r, first_laneid, active_mask);

  if (record != NULL) {
    record->address[laneid] = (uint64_t)address;
  }

  __syncwarp(active_mask);

  if (laneid == first_laneid) {
    // 5. Push a record
    gpu_queue_push(buffer);
  }

  return SANITIZER_PATCH_SUCCESS;
}


extern "C"
__device__ __noinline__
SanitizerPatchResult
sanitizer_global_memory_access_callback
(
 void *user_data,
 uint64_t pc,
 void *address,
 uint32_t size,
 uint32_t flags,
 const void *new_value
) 
{
  return memory_access_callback(user_data, pc, address, size, flags, new_value);
}


extern "C"
__device__ __noinline__
SanitizerPatchResult
sanitizer_shared_memory_access_callback
(
 void *user_data,
 uint64_t pc,
 void *address,
 uint32_t size,
 uint32_t flags,
 const void *new_value
) 
{
  return memory_access_callback(user_data, pc, address, size, flags | GPU_PATCH_SHARED, new_value);
}


extern "C"
__device__ __noinline__
SanitizerPatchResult
sanitizer_local_memory_access_callback
(
 void *user_data,
 uint64_t pc,
 void *address,
 uint32_t size,
 uint32_t flags,
 const void *new_value
) 
{
  return memory_access_callback(user_data, pc, address, size, flags | GPU_PATCH_LOCAL, new_value);
}


/*
 * Lock the corresponding hash entry for a block
 */
extern "C"
__device__ __noinline__
SanitizerPatchResult
sanitizer_block_exit_callback
(
 void *user_data,
 uint64_t pc
)
{
  gpu_patch_buffer_t* buffer = (gpu_patch_buffer_t *)user_data;

  if (!sample_callback(buffer->block_sampling_frequency, buffer->block_sampling_offset)) {
    return SANITIZER_PATCH_SUCCESS;
  }

  uint32_t active_mask = __activemask();
  uint32_t laneid = get_laneid();
  uint32_t first_laneid = __ffs(active_mask) - 1;
  int32_t pop_count = __popc(active_mask);

  if (laneid == first_laneid) {
    // Finish a bunch of threads
    atomicAdd(&buffer->num_threads, -pop_count);
  }

  return SANITIZER_PATCH_SUCCESS;
}

