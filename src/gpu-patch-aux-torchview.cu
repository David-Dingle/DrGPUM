#include "hip/hip_runtime.h"
#include "gpu-patch.h"
#include "gpu-queue.h"
#include "utils.h"

#include <sanitizer_patching.h>

struct gpu_patch_analysis_address_comparator {
  __device__
  bool operator()(gpu_patch_analysis_address &l, gpu_patch_analysis_address &r) {
    return l.start <= r.start;
  }
};

/*
 * Monitor each shared and global memory access.
 */
static 
__device__ __forceinline__
SanitizerPatchResult
memory_access_callback
(
 void *user_data,
 uint64_t pc,
 void *address,
 uint32_t size,
 uint32_t flags,
 const void *new_value
) 
{
  gpu_patch_buffer_t *buffer = (gpu_patch_buffer_t *)user_data;

  // 1. Init values
  uint32_t active_mask = __activemask();

  uint32_t byte_offset = 0;
  if (buffer->aux != NULL && (flags & (GPU_PATCH_SHARED | GPU_PATCH_LOCAL)) == 0) {
    // Read address can be filtered
    gpu_patch_aux_torchview_dict_t *address_dict = (gpu_patch_aux_torchview_dict_t *)buffer->aux;
    gpu_patch_analysis_address_t *start_end = address_dict->start_end;
    gpu_patch_analysis_address_t addr = { (uint64_t)address, 0 };
    uint32_t pos = map_prev(start_end, addr, address_dict->view_range_size, gpu_patch_analysis_address_comparator());

    if (pos != address_dict->view_range_size && (uint64_t)address < (start_end + pos)->end) { // >= start_end[pos].end >= address >= start_end[pos].start
      byte_offset = (pos / 64); // column index at corresponding (pc) row
      uint64_t bit_mask = 1 << (pos % 64); // bit offset within above byte
      uint32_t range_columns = ((address_dict->view_range_size) / 64);
      // Step 1 Find/Insert pc from/into read/write_pc_range_bit_map; get the row index
      if (static_cast<GPUPatchFlags>(flags) == GPU_PATCH_READ) {
        uint64_t* read_pc_range_map = address_dict->read_pc_range_bit_map;
        const uint64_t local_pc_offset = (pc - address_dict->function_pc_offset) / 8;
        *(read_pc_range_map + (local_pc_offset * (range_columns + 2)) + byte_offset) |= bit_mask;
      } else if (static_cast<GPUPatchFlags>(flags) == GPU_PATCH_WRITE) {
        uint64_t* write_pc_range_map = address_dict->write_pc_range_bit_map;
        const uint64_t local_pc_offset = (pc - address_dict->function_pc_offset) / 8;
        *(write_pc_range_map + (local_pc_offset * (range_columns + 2)) + byte_offset) |= bit_mask;
      } 
    } 
  }
  __syncwarp(active_mask);
  return SANITIZER_PATCH_SUCCESS;
}


extern "C"
__device__ __noinline__
SanitizerPatchResult
sanitizer_global_memory_access_callback
(
 void *user_data,
 uint64_t pc,
 void *address,
 uint32_t size,
 uint32_t flags,
 const void *new_value
) 
{
  return memory_access_callback(user_data, pc, address, size, flags, new_value);
}


extern "C"
__device__ __noinline__
SanitizerPatchResult
sanitizer_shared_memory_access_callback
(
 void *user_data,
 uint64_t pc,
 void *address,
 uint32_t size,
 uint32_t flags,
 const void *new_value
) 
{
  return memory_access_callback(user_data, pc, address, size, flags | GPU_PATCH_SHARED, new_value);
}


extern "C"
__device__ __noinline__
SanitizerPatchResult
sanitizer_local_memory_access_callback
(
 void *user_data,
 uint64_t pc,
 void *address,
 uint32_t size,
 uint32_t flags,
 const void *new_value
) 
{
  return memory_access_callback(user_data, pc, address, size, flags | GPU_PATCH_LOCAL, new_value);
}


/*
 * Lock the corresponding hash entry for a block
 */
extern "C"
__device__ __noinline__
SanitizerPatchResult
sanitizer_block_exit_callback
(
 void *user_data,
 uint64_t pc
)
{
  gpu_patch_buffer_t* buffer = (gpu_patch_buffer_t *)user_data;

  if (!sample_callback(buffer->block_sampling_frequency, buffer->block_sampling_offset)) {
    return SANITIZER_PATCH_SUCCESS;
  }

  uint32_t active_mask = __activemask();
  uint32_t laneid = get_laneid();
  uint32_t first_laneid = __ffs(active_mask) - 1;
  int32_t pop_count = __popc(active_mask);

  if (laneid == first_laneid) {
    // Finish a bunch of threads
    atomicAdd(&buffer->num_threads, -pop_count);
  }

  return SANITIZER_PATCH_SUCCESS;
}